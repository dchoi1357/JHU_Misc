
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <algorithm>
#include <cstdlib>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

// In the following section, define the model Parameters
#define N_AR 3
#define START_X 0.800, 0.900, 1.100
#define PHI -0.315415, 0.427606, 0.189134
#define C 1.500
// End model parameters

unsigned int N_SIMS, N_BLK, N_THRD, N_BYTES, T_MAX;
const unsigned int MAX_THREADS = 512; // max threads per block 
__constant__ float c_phi[N_AR]; // autoregressive parameters as constant
unsigned int N_BYTES_PARM = N_AR * sizeof(float); // size of parameter constant

// Calculate and return mean of an array of floats
float calcMean(float *arr, unsigned int const n) {
	float sum = 0.0;
	for (int i=0; i<n; i++) {
		sum += arr[i];
	}
	return sum / n; 
}

__host__ hipEvent_t get_time(void) {
	hipEvent_t time;
	hipEventCreate(&time);
	hipEventRecord(time);
	return time;
}

// Simulate a AR(n) process saving temp results to registers
__global__ void sim_register(float *x0, float *x1, float *x2, float *x_out, 
							const unsigned int N, const unsigned int T) {
	unsigned int const tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	
	if (tid < N) {
		hiprandState_t state; // initialize rand state
		hiprand_init(tid, 0, 0, &state); // set seed to thread index

		float r_x0 = x0[tid]; // Copy values of X to register
		float r_x1 = x1[tid];
		float r_x2 = x2[tid];
		float r_x; // initialize r_x
		
		float w; // white noise for AR process
		for (int t=0; t < T; t++) { // Simulate for T_MAX periods
			w = hiprand_normal(&state) / 2; // w ~ Normal(0, 0.5)
			r_x = C + c_phi[2]*r_x2 + c_phi[1]*r_x1 + c_phi[0]*r_x0 + w;
			r_x2 = r_x1;
			r_x1 = r_x0;
			r_x0 = r_x;
		}

		x_out[tid] = r_x; // save x as output
	}
}

// Simulate a AR(n) process saving work to global mem directly
__global__ void sim_gmem(float *x0, float *x1, float *x2, float *x_out, 
						const unsigned int N, const unsigned int T) {
	unsigned int const i = (blockIdx.x * blockDim.x) + threadIdx.x;
	
	if (i < N) {
		hiprandState_t state; // initialize rand state
		hiprand_init(i, 0, 0, &state); // set seed to thread index

		float w; // white noise for AR process
		for (int t=0; t < T; t++) { // Simulate for T_MAX periods
			w = hiprand_normal(&state) / 2; // w ~ Normal(0, 0.5)
			x_out[i] = C + c_phi[2]*x2[i] + c_phi[1]*x1[i] + c_phi[0]*x0[i] + w;
			x2[i] = x1[i];
			x1[i] = x0[i];
			x0[i] = x_out[i];
		}
	}
}

void simulate(const unsigned int type) {
	float *h_x0, *h_x1, *h_x2, *h_x; 
	h_x0 = (float*) malloc(N_BYTES); // allocate input
	h_x1 = (float*) malloc(N_BYTES); // allocate input
	h_x2 = (float*) malloc(N_BYTES); // allocate input
	h_x = (float*) malloc(N_BYTES); // allocate output 
		
	float start_x [N_AR] = {START_X};
	for (int i = 0; i < N_SIMS; i++) { // set all host Xs to the same number
		h_x0[i] = start_x[0];
		h_x1[i] = start_x[1];
		h_x2[i] = start_x[2];
	}
	
	float *d_x0, *d_x1, *d_x2, *d_out; // device memory for storing X
	hipMalloc((void **)&d_x0, N_BYTES); // allocate device input
	hipMalloc((void **)&d_x1, N_BYTES); // allocate device input
	hipMalloc((void **)&d_x2, N_BYTES); // allocate device input
	hipMalloc((void **)&d_out, N_BYTES); // allocate device output

	float h_phi [N_AR] = {PHI}; // constant for AR parms
	hipMemcpyToSymbol(HIP_SYMBOL(c_phi), h_phi, N_BYTES_PARM); // copy params to constant
	
	/**** Simulation *****/
	char *typeName;
	hipEvent_t start = get_time(); // start time 
	hipMemcpy(d_x0, h_x0, N_BYTES, hipMemcpyHostToDevice); //copy to device
	hipMemcpy(d_x1, h_x1, N_BYTES, hipMemcpyHostToDevice); //copy to device
	hipMemcpy(d_x2, h_x2, N_BYTES, hipMemcpyHostToDevice); //copy to device
	
	if (type == 1){ // if simulating with registers
		typeName = "registers";
		sim_register<<<N_BLK, N_THRD>>>(d_x0, d_x1, d_x2, d_out, N_SIMS, T_MAX); 
		
	} else { // if simulating with global memory 
		typeName = "global mem";
		sim_gmem<<<N_BLK, N_THRD>>>(d_x0, d_x1, d_x2, d_out, N_SIMS, T_MAX); 
	}	
	hipMemcpy(h_x, d_out, N_BYTES, hipMemcpyDeviceToHost ); // copy back
	hipEvent_t stop = get_time(); // stop time
	hipEventSynchronize(stop);
	
	// Calculate and print simulation results and timing
	float x_mu = calcMean(h_x, N_SIMS);
	float dur = 0;
	hipEventElapsedTime(&dur, start, stop);
	printf("\twith %s, result=%f, %.3f ms taken, \n", typeName, x_mu, dur);
	
	// Free up memory
	hipFree(d_x2); hipFree(d_x1); hipFree(d_x0); 
	hipFree(d_out); hipFree(c_phi);
	free(h_x0); free(h_x1); free(h_x2); free(h_x);
}

int main(int argc, char* argv[]) {
	if (argc == 3) { // get number of simulations based on CMDLINE input
		N_SIMS = atoi(argv[1]);
		T_MAX = atoi(argv[2]);
	} else {
		printf("Usage: %s [nSimulations] [maxTimePeriods].\n", argv[0]);
		return EXIT_FAILURE;
	}
	N_BLK = N_SIMS / MAX_THREADS + 1; // min of one block
	N_THRD = std::min(N_SIMS, MAX_THREADS); // num of threads per block
	N_BYTES = N_SIMS * sizeof(float); // size of array 
	printf("Running %u simulations over %u time periods...\n", N_SIMS, T_MAX);
	
	simulate(1); // simulating with registers
	simulate(2); // simulating with global memory
	return EXIT_SUCCESS;
}
