
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define MAX 100

/* this GPU kernel function calculates a random number and stores it in the parameter */
__global__ void random(float* result1, float* result2) {
    /* CUDA's random number library uses curandState_t to keep track of the seed value
       we will store a random state for every thread  */
    hiprandState_t state;

    /* we have to initialize the state */
    hiprand_init(0, /* the seed controls the sequence of random values that are produced */
            0, /* the sequence number is only important with multiple cores */
            0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
            &state);

    /* curand works like rand - except that it takes a state as a parameter */
    *result1 = hiprand_normal(&state);
	*result2 = hiprand_normal(&state);
}

int main( ) {
    /* allocate an int on the GPU */
    float* gpu_x1, * gpu_x2;
    hipMalloc((void**) &gpu_x1, sizeof(float));
	hipMalloc((void**) &gpu_x2, sizeof(float));

    /* invoke the GPU to initialize all of the random states */
    random<<<1, 1>>>(gpu_x1, gpu_x2);

    /* copy the random number back */
    float x1, x2;
    hipMemcpy(&x1, gpu_x1, sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(&x2, gpu_x2, sizeof(float), hipMemcpyDeviceToHost);

    printf("Random number = %f.\n", x1);
	printf("Random number = %f.\n", x2);

    /* free the memory we allocated */
    hipFree(gpu_x1);
	hipFree(gpu_x2);

    return 0;
}
