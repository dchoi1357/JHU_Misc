
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <algorithm>
#include <cstdlib>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

// In the following section, define the prob distribution parameters
#define N_PARAMS 3
#define PARAM1 50.0f, 3.0f, 0.5f // format: LAMBDA, A, B
#define PARAM2 1.5f, 0.8f, 5.0f

// parameters saved as constants
unsigned int N_BYTES_PRM = N_PARAMS * sizeof(float); // size of parameter 

unsigned int N_SIMS, N_BLK, N_THRD, N_BYTES_I, N_BYTES_F;
const unsigned int MAX_THREADS = 512; // max threads per block 

// Calculate and return mean of an array of floats
float calcMean(float arr[], unsigned int const n) {
	double sum = 0.0;
	for (unsigned int i=0; i<n; i++) {
		sum += (arr[i] / n);
	}
	return sum; 
}

__host__ hipEvent_t get_time(void) {
	hipEvent_t time;
	hipEventCreate(&time);
	hipEventRecord(time);
	return time;
}

__global__ void sim_freq(unsigned int *f_out, float *prm, unsigned int N) {
	unsigned int const tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	
	if (tid < N) {
		float lambda = prm[0];
		hiprandState_t state; // initialize rand state
		hiprand_init(tid, 0, 0, &state); // set seed to thread index

		f_out[tid] = hiprand_poisson(&state, lambda); // save loss frequency
	}
}

__global__ void sim_severity(float *loss_out, unsigned int *freq, float *prm,
							const unsigned int N) {
	unsigned int const tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	
	if (tid < N) {
		double A = prm[1];
		double B = prm[2];
		
		hiprandState_t state; // initialize rand state
		hiprand_init(tid, 0, 0, &state); // set seed to thread index
		double sum = 0.0;
		double unif = 0.0; // temp var for storing uniform rand 
		for (int f=0; f < freq[tid]; f++) {
			unif = hiprand_uniform_double(&state);
			sum += B / pow(1-unif, 1/A);
		}		
		loss_out[tid] = (float) sum;
	}
}

void asynch() {
	return;
}

int main(int argc, char* argv[]) {
	if (argc == 2) { // get number of simulations based on CMDLINE input
		N_SIMS = atoi(argv[1]);
	} else {
		printf("Usage: %s [nSimulations].\n", argv[0]);
		return EXIT_FAILURE;
	}
	N_BLK = N_SIMS / MAX_THREADS + 1; // min of one block
	N_THRD = std::min(N_SIMS, MAX_THREADS); // num of threads per block
	N_BYTES_F = N_SIMS * sizeof(float); // size of loss array 
	N_BYTES_I = N_SIMS * sizeof(unsigned int); // size of frequency array
	printf("Running %u simulations ...\n", N_SIMS);
	
	hipStream_t s1, s2; // Create and initialize streams
	hipStreamCreate(&s1);
	hipStreamCreate(&s2);
	
	// allocate and copy parameter to device
	float h_prm1 [N_PARAMS] = {PARAM1};
	float h_prm2 [N_PARAMS] = {PARAM2};
	hipHostRegister(h_prm1, N_BYTES_PRM, hipHostRegisterDefault);
	hipHostRegister(h_prm2, N_BYTES_PRM, hipHostRegisterDefault);
	float *d_prm1, *d_prm2;
	hipMalloc((void **)&d_prm1, N_BYTES_PRM);
	hipMalloc((void **)&d_prm2, N_BYTES_PRM);
	hipMemcpyAsync(d_prm1, h_prm1, N_BYTES_PRM, hipMemcpyHostToDevice, s1); 
	hipMemcpyAsync(d_prm2, h_prm2, N_BYTES_PRM, hipMemcpyHostToDevice, s2); 
	
	unsigned int *h_freq1, *d_freq1, *h_freq2, *d_freq2; // frequency arrays 
	float *h_loss1, *d_loss1, *h_loss2, *d_loss2; // loss arrays
	hipMalloc((void **)&d_freq1, N_BYTES_I); // device array 
	hipMalloc((void **)&d_loss1, N_BYTES_F);
	hipMalloc((void **)&d_freq2, N_BYTES_I);
	hipMalloc((void **)&d_loss2, N_BYTES_F);
	hipHostMalloc((void**)&h_freq1, N_BYTES_I, hipHostMallocDefault); // pinned host array
	hipHostMalloc((void**)&h_loss1, N_BYTES_F, hipHostMallocDefault);
	hipHostMalloc((void**)&h_freq2, N_BYTES_I, hipHostMallocDefault);
	hipHostMalloc((void**)&h_loss2, N_BYTES_F, hipHostMallocDefault);

	float dur, mean1, mean2; // to record duration and averages
	
	// ---- asynchronus run ----
	hipEvent_t start = get_time();
	hipEvent_t copyEnd1, copyEnd2;
	hipEventCreate(&copyEnd1); hipEventCreate(&copyEnd2); 
	sim_freq<<<N_BLK, N_THRD, 0, s1>>>(d_freq1, d_prm1, N_SIMS);
	sim_freq<<<N_BLK, N_THRD, 0, s2>>>(d_freq2, d_prm2, N_SIMS);
	sim_severity<<<N_BLK, N_THRD, 0, s1>>>(d_loss1, d_freq1, d_prm1, N_SIMS);
	sim_severity<<<N_BLK, N_THRD, 0, s2>>>(d_loss2, d_freq2, d_prm2, N_SIMS); 
	hipMemcpyAsync(h_loss1, d_loss1, N_BYTES_F, hipMemcpyDeviceToHost, s1);
	hipEventRecord(copyEnd1, s1);
	hipMemcpyAsync(h_loss2, d_loss2, N_BYTES_F, hipMemcpyDeviceToHost, s2);
	hipEventRecord(copyEnd2, s2);
	hipMemcpyAsync(h_freq1, d_freq1, N_BYTES_I, hipMemcpyDeviceToHost, s1);
	hipMemcpyAsync(h_freq2, d_freq2, N_BYTES_I, hipMemcpyDeviceToHost, s2);
	hipEventSynchronize(copyEnd1);
	mean1 = calcMean(h_loss1, N_SIMS);
	hipEventSynchronize(copyEnd2);
	mean2 = calcMean(h_loss2, N_SIMS);
	hipStreamSynchronize( s1 );
	hipStreamSynchronize( s2 );
	hipEvent_t stop = get_time(); // stop time
	hipEventSynchronize(stop);
	hipEventElapsedTime(&dur, start, stop);
	
	printf("\tasynchronously:\t loss1=%.3f, loss2=%.3f, %.3f ms taken, \n", 
			mean1, mean2, dur);
	
	
	// ---- synchronus run ----
	start = get_time();
	sim_freq<<<N_BLK, N_THRD>>>(d_freq1, d_prm1, N_SIMS);
	sim_severity<<<N_BLK, N_THRD>>>(d_loss1, d_freq1, d_prm1, N_SIMS);
	hipMemcpy(h_freq1, d_freq1, N_BYTES_I, hipMemcpyDeviceToHost);
	hipMemcpy(h_loss1, d_loss1, N_BYTES_F, hipMemcpyDeviceToHost);
	sim_freq<<<N_BLK, N_THRD>>>(d_freq2, d_prm2, N_SIMS);
	sim_severity<<<N_BLK, N_THRD>>>(d_loss2, d_freq2, d_prm2, N_SIMS); 
	hipMemcpy(h_freq2, d_freq2, N_BYTES_I, hipMemcpyDeviceToHost);
	hipMemcpy(h_loss2, d_loss2, N_BYTES_F, hipMemcpyDeviceToHost);
	mean1 = calcMean(h_loss1, N_SIMS);
	mean2 = calcMean(h_loss2, N_SIMS);
	stop = get_time(); // stop time
	hipEventSynchronize(stop);
	hipEventElapsedTime(&dur, start, stop);

	printf("\tsynchronously:\t loss1=%.3f, loss2=%.3f, %.3f ms taken, \n", 
			mean1, mean2, dur);
	
	return EXIT_SUCCESS;
}

