#include <stdio.h>
#include <stdlib.h>

//#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsparse.h>
#include <hipsolver.h>
#include <hipsolver.h>

unsigned int N_BYTES_MAT;

void printMatrix(int m, int n, const float *A, const char* name) {
	printf("Printing matrix %s: \n", name);
	for(int row = 0 ; row < m ; row++){
		for(int col = 0 ; col < n ; col++){
			double Areg = A[row + col*lda];
			printf("%s(%d,%d) = %f\n", name, row+1, col+1, Areg);
		}
	}
}

int main(int argc, char*argv[]) {
	hipsparseHandle_t cuSpHdl; // cuSparse handle
	hipsparseCreate(&cuSpHdl);
	
/*		| 2  0  0 |
	A =	| 0  0 -1 |
		| 3 -2  0 |
		| 0  1  0 |
	
	x = (2.0 3.0 1.0)'
	b = (4 -1 0 3)'
*/
    const int nrows = 4; // rows of matrix A
    const int ncols = 3; // rows of matrix A
	
	// Generate host side dense matrix
	float h_denseA[nrows*ncols] = {2.0, 0.0, 3.0, 0.0,   0.0, 0.0, -2.0, 1.0, 
		0.0, -1.0, 0.0, 0.0};
	float *d_denseA;
	hipMalloc((void**)&d_denseA,  sizeof(float)*nrows*ncols);
	hipMemcpy(d_denseA, h_denseA,  sizeof(float)*nrows*ncols, 
		hipMemcpyHostToDevice);
	
	// Set descriptions of sparse matrix A
	hipsparseMatDescr_t descrA;
	hipsparseCreateMatDescr(&descrA);
	hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
	
	int non0; // number of non-zero elements
	int *d_non0vec; // number of non-zero elements per row
	hipMalloc(&d_non0vec, nrows*sizeof(int));
	
	hipsparseSnnz(cuSpHdl, HIPSPARSE_DIRECTION_ROW, nrows, ncols, descrA, 
				d_denseA, nrows, d_non0vec, &non0);
				
	
	// print number of non-zero elements per row
	int *h_non0vec; 
	h_non0vec = (int*) malloc(sizeof(int)*nrows);
	hipMemcpy(h_non0vec, d_non0vec, sizeof(int)*nrows, hipMemcpyDeviceToHost); 
	for (int i=0; i < nrows; i++) {
		printf("%u ", h_non0vec[i]);
	}
	printf("\n");
	printf("Non-zero: %u\n", non0);
	
	
	// Device side sparse matrix;
	float *d_sprsA;
	int *d_rowIdx, *d_colPtr;
	hipMalloc(&d_sprsA, non0*sizeof(float));
	hipMalloc(&d_rowIdx, (nrows+1) * sizeof(int));
	hipMalloc(&d_colPtr, non0 * sizeof(int));
	hipsparseSdense2csr(cuSpHdl, nrows, ncols, descrA, d_denseA, nrows, 
					d_non0vec, d_sprsA, d_rowIdx, d_colPtr);
	
	// Get sparse matrix on the host side
	float *h_sprsA;
	int *h_rowIdx, *h_colPtr;
	h_sprsA = (float*) malloc(non0 * sizeof(float));
	h_rowIdx = (int*) malloc( (nrows+1) * sizeof(int) );
	h_colPtr = (int*) malloc( non0 * sizeof(int) );
	hipMemcpy(h_sprsA, d_sprsA, non0*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(h_rowIdx, d_rowIdx, (nrows+1)*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(h_colPtr, d_colPtr, non0*sizeof(int), hipMemcpyDeviceToHost);
	for (int i = 0; i < non0; ++i) {
		printf("sparse_A[%i] = %.0f \n", i, h_sprsA[i]);
	} 
	for (int i = 0; i < (nrows + 1); ++i) {
		printf("row_idx[%i] = %i \n", i, h_rowIdx[i]);
	}
	for (int i = 0; i < non0; ++i) {\
		printf("col_ptr[%i] = %i \n", i, h_colPtr[i]);
	}
		
	// allocate solution vector
	float h_x[ncols] = {2.0, 3.0, 1.0};
	float *d_x;
	hipMalloc(&d_x, ncols*sizeof(float));
	hipMemcpy(d_x, h_x, ncols*sizeof(float), hipMemcpyHostToDevice);
	
	// define b vector on host and device
	float *d_b, *h_b;
	hipMalloc(&d_b, nrows*sizeof(float));
	h_b = (float*) malloc(nrows*sizeof(float));
	
	float a = 1.0f; float b = 0.0f;
	hipsparseScsrmv(cuSpHdl, HIPSPARSE_OPERATION_NON_TRANSPOSE, nrows, ncols, 
		non0, &a, descrA, d_sprsA, d_rowIdx, d_colPtr, d_x, &b, d_b);
	hipMemcpy(h_b, d_b, nrows*sizeof(float), hipMemcpyDeviceToHost);
	
	for (int i=0; i < nrows; i++) {
		printf("b[%u] = %f", i, h_b[i]);
	}

	printf("Good!\n");
	return EXIT_SUCCESS;
}
