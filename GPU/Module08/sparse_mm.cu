#include <stdio.h>
#include <stdlib.h>

//#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsparse.h>
#include <hipsolver.h>
#include <hipsolver.h>

// look into hipsparseXcsrgemmNnz 

unsigned int N_BYTES_MAT;

void printMatrix(int m, int n, const double*A, int lda, const char* name)
{
	for(int row = 0 ; row < m ; row++){
		for(int col = 0 ; col < n ; col++){
			double Areg = A[row + col*lda];
			printf("%s(%d,%d) = %f\n", name, row+1, col+1, Areg);
		}
	}
}


int main(int argc, char*argv[]) {
	hipsparseHandle_t cuSpHdl; // cuSparse handle
	hipsparseCreate(&cuSpHdl);
	
/*  	| 2  0  0 |
	A = | 0  0 -1 |
		| 3 -2  0 |
		| 0  1  0 |
	
	x = (2.1034 2.7241 1.0000)'
	b = (4 -1 1 3)'
*/
    const int nrows = 4; // rows of matrix A
    const int ncols = 3; // rows of matrix A
	N_BYTES_MAT = sizeof(float)*nrows*ncols;
	
	// Generate host side dense matrix
	float h_denseA[nrows*ncols] = {2.0, 0.0, 3.0, 0.0,   0.0, 0.0, -2.0, 1.0, 
		0.0, -1.0, 0.0, 0.0};
	float *d_denseA;
	hipMalloc((void**)&d_denseA, N_BYTES_MAT);
	hipMemcpy(d_denseA, h_denseA, N_BYTES_MAT, hipMemcpyHostToDevice);
	
	// Set descriptions of sparse matrix A
	hipsparseMatDescr_t descrA;
	hipsparseCreateMatDescr(&descrA);
	hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
	
	int non0; // number of non-zero elements
	int *d_non0vec; // number of non-zero elements per row
	hipMalloc(&d_non0vec, nrows*sizeof(int));
	
	hipsparseSnnz(cuSpHdl, HIPSPARSE_DIRECTION_ROW, nrows, ncols, descrA, 
				d_denseA, nrows, d_non0vec, &non0);
				
	
	// print number of non-zero elements per row
	int *h_non0vec; 
	h_non0vec = (int*) malloc(sizeof(int)*nrows);
	hipMemcpy(h_non0vec, d_non0vec, sizeof(int)*nrows, hipMemcpyDeviceToHost); 
	for (int i=0; i < nrows; i++) {
		printf("%u ", h_non0vec[i]);
	}
	printf("\n");
	printf("Non-zero: %u\n", non0);
	
	
	// Device side sparse matrix;
	float *d_sprsA;
	int *d_rowIdx, *d_colPtr;
	hipMalloc(&d_sprsA, non0*sizeof(float));
	hipMalloc(&d_rowIdx, (nrows+1) * sizeof(int));
	hipMalloc(&d_colPtr, non0 * sizeof(int));
	hipsparseSdense2csr(cuSpHdl, nrows, ncols, descrA, d_denseA, nrows, 
					d_non0vec, d_sprsA, d_rowIdx, d_colPtr);
	
	// Get sparse matrix on the host side
	float *h_sprsA;
	int *h_rowIdx, *h_colPtr;
	h_sprsA = (float*) malloc(non0 * sizeof(float));
	h_rowIdx = (int*) malloc( (nrows+1) * sizeof(int) );
	h_colPtr = (int*) malloc( non0 * sizeof(int) );
	hipMemcpy(h_sprsA, d_sprsA, non0*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(h_rowIdx, d_rowIdx, (nrows+1)*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(h_colPtr, d_colPtr, non0*sizeof(int), hipMemcpyDeviceToHost);
	for (int i = 0; i < non0; ++i) {
		printf("sparse_A[%i] = %.0f \n", i, h_sprsA[i]);
	} 
	for (int i = 0; i < (nrows + 1); ++i) {
		printf("row_idx[%i] = %i \n", i, h_rowIdx[i]);
	}
	for (int i = 0; i < non0; ++i) {\
		printf("col_ptr[%i] = %i \n", i, h_colPtr[i]);
	}
	
	// define b vector on host and device
	float h_b[nrows] = {4.0, -1.0, 1.0, 3.0};
	float *d_b;
	hipMalloc(&d_b, nrows*sizeof(float));
	hipMemcpy(d_b, h_b, nrows*sizeof(float), hipMemcpyHostToDevice);
	
	// allocate solution vector
	float *d_x, *h_x;
	hipMalloc(&d_x, ncols*sizeof(float));
	h_x = (float*) malloc(ncols*sizeof(float));
	
	
	// initialize cuSolver
	hipsolverSpHandle_t cuSolvHdl;
    hipsolverSpCreate(&cuSolvHdl);
	int *d_p, *h_p;
	hipMalloc(&d_p, ncols*sizeof(int));
	h_p = (int*) malloc(ncols*sizeof(int));
	int rankA;
	float minNorm;
	
	/*
	cusolverSpScsrlsqvqr(cuSolvHdl, nrows, ncols, non0, descrA, d_sprsA, 
		d_rowIdx, d_colPtr, d_b, 1e-6, rankA, d_x, d_p, &minNorm);
	*/
	cusolverSpScsrlsqvqrHost(cuSolvHdl, nrows, ncols, non0, descrA, h_sprsA, 
		h_rowIdx, h_colPtr, h_b, 1e-6, &rankA, h_x, h_p, &minNorm);
	
	printf("Good!\n");
	return EXIT_SUCCESS;
}
